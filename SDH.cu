/* ==================================================================
	Programmer: Yicheng Tu (ytu@cse.usf.edu)
	The basic SDH algorithm implementation for 3D data
	To compile: nvcc SDH.c -o SDH in the C4 lab machines
   ==================================================================
*/

/* USF Fall 2019 CIS4930 Programming on Massively Parallel Systems
   Project Description: Write a CUDA program to implement the same
   functionality as the CPU only code

   Student: Alexander Cook
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>


#define BOX_SIZE	23000 /* size of the data box on one dimension            */

/* descriptors for single atom in the tree */
typedef struct atomdesc {
	double x_pos;
	double y_pos;
	double z_pos;
} atom;

typedef struct hist_entry{
	//float min;
	//float max;
	unsigned long long d_cnt;   /* need a long long type as the count might be huge */
} bucket;

//Global variables
bucket * h_histogram;	/* list of all buckets in the histogram     */
long long	PDH_acnt;	/* total number of data points              */
int num_buckets;		/* total number of buckets in the histogram */
double   PDH_res;		/* value of w                               */
atom * atom_list;		/* list of all data points					*/

//Device helper function: Distance of two points in the atom_list
//Device helper function which now takes a pointer as an argument instead of using a global pointer
__device__ double d_p2p_distance(atom *atom_list, int ind1, int ind2) {
	
	double x1 = atom_list[ind1].x_pos;
	double x2 = atom_list[ind2].x_pos;
	double y1 = atom_list[ind1].y_pos;
	double y2 = atom_list[ind2].y_pos;
	double z1 = atom_list[ind1].z_pos;
	double z2 = atom_list[ind2].z_pos;
		
	return sqrt((x1 - x2)*(x1-x2) + (y1 - y2)*(y1 - y2) + (z1 - z2)*(z1 - z2));
}


__global__ void PDH_kernel(atom *d_atom_list, bucket *d_histogram, int PDH_acnt, int PDH_res, int num_buckets){
	int T = blockIdx.x * blockDim.x + threadIdx.x;
	int t = threadIdx.x;

	extern __shared__ int s_histogram[];

	for(int i = t; i < num_buckets; i += blockDim.x){ //initialize array to 0 in block sized chunks
		s_histogram[i] = 0;
	}

	__syncthreads();
 
	int h_pos;
	double dist;

	for(int i = T + 1; i < PDH_acnt; i++){
		dist = d_p2p_distance(d_atom_list, T, i);
			h_pos = (int) (dist / PDH_res);
			atomicAdd(&(s_histogram[h_pos]), 1);
			 
	}

	__syncthreads();

	//reduce shared output into global output
	for(int i = t; i < num_buckets; i += blockDim.x){ //output to global memory in block sized chunks
		atomicAdd(&(d_histogram[i].d_cnt), s_histogram[i]);
	}

}


/* 
	Print the counts in all buckets of the histogram 
*/
void output_histogram(bucket *histogram){
	int i; 
	long long total_cnt = 0;
	for(i=0; i< num_buckets; i++) {
		if(i%5 == 0) /* we print 5 buckets in a row */
			printf("\n%02d: ", i);
		printf("%15lld ", histogram[i].d_cnt);
		total_cnt += histogram[i].d_cnt;
	  	/* we also want to make sure the total distance count is correct */
		if(i == num_buckets - 1)	
			printf("\n T:%lld \n", total_cnt);
		else printf("| ");
	}
}


int main(int argc, char **argv)
{
	int i;

	PDH_acnt = atoi(argv[1]);
	PDH_res	 = atof(argv[2]);

	//Allocate host memory
	num_buckets = (int)(BOX_SIZE * 1.732 / PDH_res) + 1;
	h_histogram = (bucket *)malloc(sizeof(bucket) * num_buckets);
	atom_list = (atom *)malloc(sizeof(atom)*PDH_acnt);

	//initialize histogram to zero
	memset(h_histogram, 0, sizeof(bucket)*num_buckets);

	//Allocate device memory
	bucket *d_histogram; //pointer to array of buckets
	atom *d_atom_list; //pointer to array of atoms

	hipMalloc((void**)&d_histogram, sizeof(bucket)*num_buckets);
	hipMalloc((void**)&d_atom_list, sizeof(atom)*PDH_acnt);
	
	srand(1);
	/* Generate data following a uniform distribution */
	for(i = 0;  i < PDH_acnt; i++) {
		atom_list[i].x_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
		atom_list[i].y_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
		atom_list[i].z_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
	}

	//Copy host data to device memory
	hipMemcpy(d_histogram, h_histogram, sizeof(bucket)*num_buckets, hipMemcpyHostToDevice);
	hipMemcpy(d_atom_list, atom_list, sizeof(atom)*PDH_acnt, hipMemcpyHostToDevice);

	//Define block and grid size
	int num_threads = 256; //number of threads in one dimension of a block
	int num_blocks = (PDH_acnt + num_threads - 1)/num_threads; //calculate number of blocks needed
	
	//Start counting time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	//Launch kernel
	PDH_kernel<<<num_blocks,num_threads, sizeof(bucket)*num_buckets>>>(d_atom_list, d_histogram, PDH_acnt, PDH_res, num_buckets);
	//PDH_kernelST<<<1,1>>>(d_atom_list, d_histogram, PDH_acnt, PDH_res);

	//stop counting time
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	

	//Copy data from gpu memory to host memory
	hipMemcpy(h_histogram, d_histogram, sizeof(bucket)*num_buckets, hipMemcpyDeviceToHost);
	
	/* Print out the histogram again for gpu version */
	output_histogram(h_histogram);

	//report running time
	printf("******** Total Running Time of Kernel = %0.5f ms *******\n", elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	free(h_histogram);
	hipFree(d_histogram);
	
	

	return 0;
}


